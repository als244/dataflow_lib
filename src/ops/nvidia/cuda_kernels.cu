#include "hip/hip_runtime.h"
#include "norm/rms_norm.cu"
#include "norm/rms_norm_bwd_x.cu"
#include "norm/rms_norm_bwd_w.cu"

#include "attention/rope.cu"
#include "attention/copy_to_seq_context.cu"

#include "moe/select_experts.cu"

#include "activations/swiglu.cu"
#include "activations/swiglu_bwd.cu"

#include "loss/softmax.cu"
#include "loss/cross_entropy.cu"
